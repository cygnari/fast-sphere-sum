#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <array>
#include <vector>
// #include <Accelerate/Accelerate.h>
#include <fstream>
#include <sstream>
#include <chrono>
#include <cassert>

#include "helpers_cuda.h"
#include "helpers.h"

// time taken: 103913287 microseconds / 103.9 seconds for 2562 particles, 100 time steps, flattened + O3

using namespace std;

__global__ void BVE_ffunc(double* modify, double* curr_state, double t, double delta_t, double omega, double area, int points) {
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id < points) {
        // vector<double> pos_change {0, 0, 0};
        // double[3] pos_change = {0, 0, 0};
        double pos_change[3] = {0, 0, 0};
        // vector<double> particle_i {0, 0, 0};
        // double[3] particle_i;
        double particle_i[3];
        slice_cuda(curr_state, particle_i, 4 * id, 1, 3);
        for (int j = 0; j < points; j++) {
            if (id != j) {
                // vector<double> particle_j = slice2(curr_state, 4 * j, 1, 3);
                // double[3] particle_j;
                double particle_j[3];
                // vector<double> particle_j {0, 0, 0};
                slice_cuda(curr_state, particle_j, 4 * j, 1, 3);
                // double[3] contribution;
                double contribution[3];
                BVE_gfunc_cuda(particle_i, particle_j, contribution);
                scalar_mult_cuda_dev(contribution, curr_state[4 * j + 3] * area, 3);
                vec_add_cuda_dev(pos_change, contribution, 3);
            }
        }
        scalar_mult_cuda_dev(pos_change, -1.0 / (4.0 * M_PI), 3);
        for (int j = 0; j < 3; j++) modify[4 * id + j] = pos_change[j];
        modify[4 * id + 3] = -2 * omega * pos_change[2];
    }
}

__global__ void projection(double* curr_state, double radius, int points) {
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id < points) {
        // vector<double> projected {0, 0, 0};
        // double[3] projected;
        double projected[3];
        slice_cuda(curr_state, projected, 4 * id, 1, 3);
        project_to_sphere_cuda(projected, radius);
        for (int i = 0; i < 3; i++) curr_state[4 * id + i] = projected[i];
    }
}

#define point_count 655362

int main() {
    double delta_t = 0.01, end_t = 1;
    double omega = 2 * M_PI;
    int time_steps = end_t / delta_t;
    double area = (4 * M_PI) / point_count;

    vector<double> curr_state(4 * point_count); // 0 is x_pos, 1 is y_pos, 2 is z_pos, 3 is vorticity
    vector<double> c_1(4 * point_count, 0);
    vector<double> c_2(4 * point_count, 0);
    vector<double> c_3(4 * point_count, 0);
    vector<double> c_4(4 * point_count, 0);
    vector<double> c1234(4 * point_count, 0);
    vector<double> intermediate_1(4 * point_count);
    vector<double> intermediate_2(4 * point_count);
    vector<double> intermediate_3(4 * point_count);

    // fstream file("../points.csv");
    fstream file("./points.csv");
    string line, word;

    ofstream write_out;
    write_out.open("direct_output.csv", ofstream::out | ofstream::trunc);

    for (int i = 0; i < point_count; i++) {
        getline(file, line);
        stringstream str(line);
        for (int j = 0; j < 4; j++) {
            getline(str, word, ',');
            curr_state[4 * i + j] = stod(word);
        }
    }

    double *d_curr, *d_c1, *d_c2, *d_c3, *d_c4, *d_c1234, *d_inter1, *d_inter2, *d_inter3;

    hipMalloc(&d_curr, point_count*sizeof(double));
    hipMalloc(&d_c1, point_count*sizeof(double));
    hipMalloc(&d_c2, point_count*sizeof(double));
    hipMalloc(&d_c3, point_count*sizeof(double));
    hipMalloc(&d_c4, point_count*sizeof(double));
    hipMalloc(&d_c1234, point_count*sizeof(double));
    hipMalloc(&d_inter1, point_count*sizeof(double));
    hipMalloc(&d_inter2, point_count*sizeof(double));
    hipMalloc(&d_inter3, point_count*sizeof(double));

    hipMemcpy(d_curr, &curr_state[0], point_count*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c1, &c_1[0], point_count*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c2, &c_2[0], point_count*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c3, &c_3[0], point_count*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c4, &c_4[0], point_count*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c1234, &c1234[0], point_count*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_inter1, &intermediate_1[0], point_count*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_inter2, &intermediate_2[0], point_count*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_inter3, &intermediate_3[0], point_count*sizeof(double), hipMemcpyHostToDevice);

    chrono::steady_clock::time_point begin = chrono::steady_clock::now();

    // write out initial state
    // for (int i = 0; i < point_count; i++) {
    //     write_out << curr_state[4 * i] << "," << curr_state[4 * i + 1] << "," << curr_state[4 * i + 2] << "," << curr_state[4 * i + 3] << "\n";
    // }

    for (int t = 0; t < 1; t++) { // time iterate with RK4
        double curr_time = t * delta_t;
        hipDeviceSynchronize();
        BVE_ffunc<<<(point_count + 255) / 256, 256>>>(d_c1, d_curr, curr_time, delta_t, omega, area, point_count);
        hipDeviceSynchronize();
        copy_cuda<<<(point_count + 255) / 256, 256>>>(d_inter1, d_c1, point_count);
        hipDeviceSynchronize();
        scalar_mult_cuda<<<(point_count + 255) / 256, 256>>>(d_inter1, delta_t / 2, point_count);
        hipDeviceSynchronize();
        vec_add_cuda<<<(point_count + 255) / 256, 256>>>(d_inter1, d_curr, point_count);
        hipDeviceSynchronize();
        BVE_ffunc<<<(point_count + 255) / 256, 256>>>(d_c2, d_inter1, curr_time + delta_t / 2, delta_t, omega, area, point_count);
        hipDeviceSynchronize();
        copy_cuda<<<(point_count + 255) / 256, 256>>>(d_inter2, d_c2, point_count);
        hipDeviceSynchronize();
        scalar_mult_cuda<<<(point_count + 255) / 256, 256>>>(d_inter2, delta_t / 2, point_count);
        hipDeviceSynchronize();
        vec_add_cuda<<<(point_count + 255) / 256, 256>>>(d_inter2, d_curr, point_count);
        hipDeviceSynchronize();
        BVE_ffunc<<<(point_count + 255) / 256, 256>>>(d_c3, d_inter2, curr_time + delta_t / 2, delta_t, omega, area, point_count);
        hipDeviceSynchronize();
        copy_cuda<<<(point_count + 255) / 256, 256>>>(d_inter3, d_c3, point_count);
        hipDeviceSynchronize();
        scalar_mult_cuda<<<(point_count + 255) / 256, 256>>>(d_inter3, delta_t, point_count);
        hipDeviceSynchronize();
        vec_add_cuda<<<(point_count + 255) / 256, 256>>>(d_inter3, d_curr, point_count);
        hipDeviceSynchronize();
        BVE_ffunc<<<(point_count + 255) / 256, 256>>>(d_c4, d_inter3, curr_time + delta_t, delta_t, omega, area, point_count);
        hipDeviceSynchronize();
        copy_cuda<<<(point_count + 255) / 256, 256>>>(d_c1234, d_c1, point_count);
        hipDeviceSynchronize();
        scalar_mult_cuda<<<(point_count + 255) / 256, 256>>>(d_c2, 2, point_count);
        hipDeviceSynchronize();
        vec_add_cuda<<<(point_count + 255) / 256, 256>>>(d_c1234, d_c2, point_count);
        hipDeviceSynchronize();
        scalar_mult_cuda<<<(point_count + 255) / 256, 256>>>(d_c3, 2, point_count);
        hipDeviceSynchronize();
        vec_add_cuda<<<(point_count + 255) / 256, 256>>>(d_c1234, d_c3, point_count);
        hipDeviceSynchronize();
        vec_add_cuda<<<(point_count + 255) / 256, 256>>>(d_c1234, d_c4, point_count);
        hipDeviceSynchronize();
        scalar_mult_cuda<<<(point_count + 255) / 256, 256>>>(d_c1234, delta_t / 6, point_count);
        hipDeviceSynchronize();
        vec_add_cuda<<<(point_count + 255) / 256, 256>>>(d_curr, d_c1234, point_count);
        hipDeviceSynchronize();
        projection<<<(point_count + 255) / 256, 256>>>(d_curr, 1.0, point_count);
        // for (int i = 0; i < point_count; i++) {
        //     vector<double> projected = slice(curr_state, 4 * i, 1, 3);
        //     // projected = project_to_sphere(projected, 1);
        //     project_to_sphere(projected, 1);
        //     for (int j = 0; j < 3; j++) curr_state[4 * i + j] = projected[j]; // reproject points to surface of sphere
        //     // write_out << curr_state[4 * i] << "," << curr_state[4 * i + 1] << "," << curr_state[4 * i + 2] << "," << curr_state[4 * i + 3] << "\n"; // write position
        // }
        // cout << t << endl;
    }

    chrono::steady_clock::time_point end = chrono::steady_clock::now();
    cout << "time taken: " << chrono::duration_cast<chrono::microseconds>(end - begin).count() << " microseconds" << endl;

    hipFree(d_curr);
    hipFree(d_c1);
    hipFree(d_c2);
    hipFree(d_c3);
    hipFree(d_c4);
    hipFree(d_c1234);
    hipFree(d_inter1);
    hipFree(d_inter2);
    hipFree(d_inter3);

    write_out.close();
    return 0;
}
